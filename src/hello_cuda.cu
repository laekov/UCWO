#include "ucwo.hh"
#include <stdlib.h>
#include <unistd.h>
#include <mpi.h>
#include <assert.h>

int main() {
    MPI_Init(0, 0);
    int rank;
    MPI_Comm_rank(MPI_COMM_WORLD, &rank);
    UCWO::World world(MPI_COMM_WORLD);
    char* x;
    hipMalloc(&x, 1024);
    (char*)world.expose(x, 1024, UCS_MEMORY_TYPE_CUDA);
    MPI_Barrier(MPI_COMM_WORLD);

    auto w = world.newWorker();

    char data[10];
    data[0] = 'a' + rank;
    w->put(rank ^ 1, 0, 0, data, 1).wait();

    MPI_Barrier(MPI_COMM_WORLD);

    char y[1024];
    hipMemcpy(y, x, 1024, hipMemcpyDeviceToHost);
    fprintf(stderr, "Rank %d got %c\n", rank, y[0]);

    MPI_Finalize();
}

#include "ucwo.hh"
#include <stdlib.h>
#include <unistd.h>
#include <mpi.h>
#include <assert.h>

int main() {
    MPI_Init(0, 0);
    int rank;
    MPI_Comm_rank(MPI_COMM_WORLD, &rank);
    UCWO::World world(MPI_COMM_WORLD);
    char *x, *h;
    hipMalloc(&x, 1024);
    world.expose(x, 1024, UCS_MEMORY_TYPE_CUDA);
    // x = (char*)world.expose(0, 1024, UCS_MEMORY_TYPE_CUDA);
    h = (char*)world.expose(0, 1024);
    h[0] = 'A' + rank;
    // MPI_Barrier(MPI_COMM_WORLD);

    auto w = world.newWorker();

    char data[10];
    char y[1024];
    data[0] = 'a' + rank;
    w->put(rank ^ 1, 0, 0, data, 1).wait();
    w->get(rank ^ 1, 1, 0, x + 1, 1).wait();
    // w->get(rank ^ 1, 1, 0, y, 1).wait();

    MPI_Barrier(MPI_COMM_WORLD);

    hipMemcpy(y, x, 1024, hipMemcpyDeviceToHost);
    fprintf(stderr, "Rank %d got %c %c\n", rank, y[0], y[1]);

    MPI_Finalize();
}
